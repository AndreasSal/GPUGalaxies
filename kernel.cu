#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <cstdlib>
#include <stdio.h>
#include <math.h>
#include <chrono> 



__global__ void CalcDD(float* realAnglesArray, int* DD, unsigned long long int *counter, int N)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < N) {

		int j;
		float pi = atanf(1) * 4;
		float angleGalaxies;
		float alpha1;
		float alpha2;
		float delta1;
		float delta2;
		float x;
		int arrayIndex;


		//D
		alpha1 = realAnglesArray[2 * i] * pi / (60 * 180);
		delta1 = realAnglesArray[2 * i + 1] * pi / (60 * 180);

		float sinfDelta1 = sinf(delta1);
		float cosfDelta1 = cosf(delta1);

		//Histogram DD:
		for (j = 0; j < N; j++) {
			//D
			alpha2 = realAnglesArray[2 * j] * pi / (60 * 180);
			delta2 = realAnglesArray[2 * j + 1] * pi / (60 * 180);

			//angle between two galaxies a1d1 a2d2 in degrees
			// x is used to make sure that precision errors never cause acosf() to return NaN if x>1
			x = sinfDelta1 * sinf(delta2) + cosfDelta1 * cosf(delta2) * cosf(alpha1 - alpha2);
			if (x > 1) {
				x = 1;
			}
			if (x < -1) {
				x = -1;
			}
			angleGalaxies = acosf(x) * 180 / pi;
			arrayIndex = 4 * angleGalaxies;
			//adds to counter for every addition to histogram
			atomicAdd(counter, 1);
			atomicAdd(&DD[arrayIndex], 1);
		}
	}
}
__global__ void CalcRR(float* syntheticAnglesArray, int* RR, unsigned long long int *counter, int N)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < N) {

		int j;
		float pi = atanf(1) * 4;
		float angleGalaxies;
		float alpha1;
		float alpha2;
		float delta1;
		float delta2;
		float x;
		int arrayIndex;


		//R
		alpha1 = syntheticAnglesArray[2 * i] * pi / (60 * 180);
		delta1 = syntheticAnglesArray[2 * i + 1] * pi / (60 * 180);

		float sinfDelta1 = sinf(delta1);
		float cosfDelta1 = cosf(delta1);

		//Histogram RR:
		for (j = 0; j < N; j++) {
			//R
			alpha2 = syntheticAnglesArray[2 * j] * pi / (60 * 180);
			delta2 = syntheticAnglesArray[2 * j + 1] * pi / (60 * 180);

			//angle between two galaxies a1d1 a2d2 in degrees
			// x is used to make sure that precision errors never cause acosf() to return NaN if x>1
			x = sinfDelta1 * sinf(delta2) + cosfDelta1 * cosf(delta2) * cosf(alpha1 - alpha2);
			if (x > 1) {
				x = 1;
			}
			if (x < -1) {
				x = -1;
			}
			angleGalaxies = acosf(x) * 180 / pi;
			arrayIndex = 4 * angleGalaxies;
			//adds to counter for every addition to histogram
			atomicAdd(counter, 1);
			atomicAdd(&RR[arrayIndex], 1);
		}
	}
}

__global__ void CalcDR(float* realAnglesArray, float* syntheticAnglesArray, int* DR, unsigned long long int *counter, int N)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < N) {

		int j;
		float pi = atanf(1) * 4;
		float angleGalaxies;
		float alpha1;
		float alpha2;
		float delta1;
		float delta2;
		float x;
		int arrayIndex;


		//D
		alpha1 = realAnglesArray[2 * i] * pi / (60 * 180);
		delta1 = realAnglesArray[2 * i + 1] * pi / (60 * 180);

		float sinfDelta1 = sinf(delta1);
		float cosfDelta1 = cosf(delta1);

		//Histogram DR:
		for (j = 0; j < N; j++) {
			//R
			alpha2 = syntheticAnglesArray[2 * j] * pi / (60 * 180);
			delta2 = syntheticAnglesArray[2 * j + 1] * pi / (60 * 180);

			//angle between two galaxies a1d1 a2d2 in degrees
			// x is used to make sure that precision errors never cause acosf() to return NaN if x>1
			x = sinfDelta1 * sinf(delta2) + cosfDelta1 * cosf(delta2) * cosf(alpha1 - alpha2);
			if (x > 1) {
				x = 1;
			}
			if (x < -1) {
				x = -1;
			}
			angleGalaxies = acosf(x) * 180 / pi;
			arrayIndex = 4 * angleGalaxies;
			//adds to counter for every addition to histogram
			atomicAdd(counter, 1);
			atomicAdd(&DR[arrayIndex], 1);
		}
	}
}


int main()

{	
	//timer to measure performance
	using namespace std::chrono;

	auto start = high_resolution_clock::now();

	//number of threads for histogram calculation (N)
	const int N = 100000;
	//each galaxy has 2 angles
	const int angleAmount = 2*N;
	int i;
	float pi = atanf(1) * 4;
	int histogramIndex;


	//Variables for calculating statistics
	float DDf;
	float DRf;
	float RRf;
	float omega[360];

	float *realAnglesArray;
	float *syntheticAnglesArray;

	//histograms 90 DEG, 0.25 spacing
	int *DD;
	int *RR;
	int *DR;


	//counter to keep track of the number of histogram additions (10 000 000 000 x 3 if N = 100 000)
	unsigned long long int *counter;


	size_t arraybytes = angleAmount * sizeof(float);
	size_t harraybytes = 360 * sizeof(int);
	size_t counterbytes = sizeof(unsigned long long int);

	// Allocate using unified pointers
	hipMallocManaged((void **)&realAnglesArray, arraybytes);
	hipMallocManaged((void **)&syntheticAnglesArray, arraybytes);
	hipMallocManaged((void **)&DD, harraybytes);
	hipMallocManaged((void **)&RR, harraybytes);
	hipMallocManaged((void **)&DR, harraybytes);
	hipMallocManaged((void **)&counter, counterbytes);
	*counter = 0;

	// initialize arrays DD, RR, DR
	for (int i = 0; i < 360; i++) {
		DD[i] = 0;
		RR[i] = 0;
		DR[i] = 0;
	}

	printf("Reading data from files:\n\n");
	FILE *angles;
	angles = fopen("flat_100k_arcmin.txt", "r");

	//read synthetic data file into array. 200 000 length
	if (angles == NULL) {
		printf("Error Reading File\n");
		exit(0);
	}

	for (i = 0; i < angleAmount; i++) {
		fscanf(angles, "%f", &syntheticAnglesArray[i]);
	}

	fclose(angles);

	angles = fopen("data_100k_arcmin.txt", "r");

	//read real data file into array. 200 000 length
	if (angles == NULL) {
		printf("Error Reading File\n");
		exit(0);
	}

	for (i = 0; i < angleAmount; i++) {
		fscanf(angles, "%f", &realAnglesArray[i]);
	}

	fclose(angles);	


	// Invoke kernel
	int threadsInBlock = 256;
	int blocksInGrid = (N + threadsInBlock - 1) / threadsInBlock;
	
	printf("\nCalculating DD...");
	CalcDD << <blocksInGrid, threadsInBlock >> > (realAnglesArray, DD, counter, N);
	hipDeviceSynchronize();
	printf("\nDone. Counter (number of additions): %llu", *counter);

	printf("\nCalculating RR...");
	CalcRR << <blocksInGrid, threadsInBlock >> > (syntheticAnglesArray, RR, counter, N);
	hipDeviceSynchronize();
	printf("\nDone. Counter (number of additions): %llu", *counter);

	printf("\nCalculating DR...");
	CalcDR << <blocksInGrid, threadsInBlock >> > (realAnglesArray, syntheticAnglesArray, DR, counter, N);
	hipDeviceSynchronize();
	printf("\nDone. Counter (number of additions): %llu", *counter);

	//print results
	for (i = 0; i < 360; i++) {
		histogramIndex = i / 4;
		printf("\nHistogram DD: i: %d  Value: %d", histogramIndex, DD[i]);
	}
	for (i = 0; i < 360; i++) {
		histogramIndex = i / 4;
		printf("\nHistogram RR: i: %d  Value: %d", histogramIndex, RR[i]);
	}
	for (i = 0; i < 360; i++) {
		histogramIndex = i / 4;
		printf("\nHistogram DR: i: %d  Value: %d", histogramIndex, DR[i]);
	}

	//calculating statistics
	for (i = 0; i < 360; i++) {
		DDf = DD[i];
		DRf = DR[i];
		RRf = RR[i];
		omega[i] = (DDf - 2 * DRf + RRf) / RRf;
		printf("\nOmega: i: %d  Value: %f", i, omega[i]);
	}


	//stop timer 
	auto stop = high_resolution_clock::now();
	auto duration = duration_cast<microseconds>(stop - start);

	printf("\nRun time (ms): %d", duration.count()/1000);

	return 0;
}

/*
19.2.2020:
threadsInBlock: 512
Run time (ms): 24610, 21200, 21615
threadsInBlock: 256
Run time (ms): 21668, 21138

threadsInBlock: 256, no atomicadd counter:
Run time (ms): 17093, 16956, 17171

Last values of DR: 
Histogram DR: i: 88  Value: 715978
Histogram DR: i: 89  Value: 528683
Histogram DR: i: 89  Value: 363505
Histogram DR: i: 89  Value: 214580
Histogram DR: i: 89  Value: 69851
*/